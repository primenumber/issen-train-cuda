#include "hip/hip_runtime.h"
#include <algorithm>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <optional>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "bitboard.hpp"
#include "sparse_mat.hpp"

struct Param {
  std::string input_path, config_path, output_path;
  size_t from, to, width;
};

Param parse_options(int argc, char** argv) {
  if (argc < 7) {
    std::cerr << "Usage: " << argv[0] << " from to width config_path input_path output_path" << std::endl;
    exit(EXIT_FAILURE);
  }
  const size_t from = std::stoi(argv[1]);
  const size_t to = std::stoi(argv[2]);
  const size_t width = std::stoi(argv[3]);
  const std::string config_path = argv[4];
  const std::string input_path = argv[5];
  const std::string output_path = argv[6];
  return {
    input_path, config_path, output_path,
    from, to, width,
  };
}

struct Config {
  size_t stones_from, stones_to;
  std::vector<uint64_t> masks;
};

Config load_config(const std::string& config_path) {
  std::ifstream ifs(config_path);
  size_t stones_from, stones_to;
  ifs >> stones_from >> stones_to;
  size_t mask_count;
  ifs >> mask_count;
  std::string mask_str;
  std::vector<uint64_t> masks;
  for (size_t i = 0; i < mask_count; ++i) {
    ifs >> mask_str;
    uint64_t mask = 0;
    for (size_t j = 0; j < 64; ++j) {
      if (mask_str[j] == '1') {
        mask |= 1;
      }
      mask <<= 1;
    }
    masks.push_back(mask);
  }
  return {
    stones_from, stones_to,
    masks
  };
}

DataSet load_dataset(const std::string& input_path) {
  std::ifstream ifs(input_path);
  size_t length;
  ifs >> length;
  DataSet result;
  result.reserve(length);
  for (size_t i = 0; i < length; ++i) {
    uint64_t player, opponent;
    int32_t score;
    uint32_t best_pos;
    ifs >> std::hex >> player >> opponent >> std::dec >> score >> best_pos;
    result.push_back({player, opponent, score});
  }
  return result;
}

#define CHECK_CUDA(expr) \
{ \
  hipError_t status = (expr); \
  if (status != hipSuccess) { \
    std::cerr << "CUDA API failed at line: " << __LINE__ << " with error: " << hipGetErrorString(status) << " (" << status << ")" << std::endl; \
    std::exit(EXIT_FAILURE); \
  } \
}

#define CHECK_CUSPARSE(expr) \
{ \
  hipsparseStatus_t status = (expr); \
  if (status != HIPSPARSE_STATUS_SUCCESS) { \
    std::cerr << "CUSPARSE API failed at line: " << __LINE__ << " with error: " << hipsparseGetErrorString(status) << "-" << hipsparseGetErrorName(status) << " (" << status << ")" << std::endl; \
    std::exit(EXIT_FAILURE); \
  } \
}

class Handle {
 public:
  Handle() {
    CHECK_CUSPARSE(hipsparseCreate(&handle))
  }
  ~Handle() { hipsparseDestroy(handle); }
  const hipsparseHandle_t& get() const { return handle; }
  hipsparseHandle_t& get() { return handle; }
 private:
  hipsparseHandle_t handle;
};

class Stream {
 public:
  Stream() {
    CHECK_CUDA(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking))
  }
  ~Stream() { hipStreamDestroy(stream); }
  const hipStream_t& get() const { return stream; }
  hipStream_t& get() { return stream; }
 private:
  hipStream_t stream;
};

struct Context {
  Handle handle;
  Stream stream;
};

struct CSRMatDev {
  explicit CSRMatDev(const CSRMat& mat) {
    CHECK_CUDA(hipMalloc((void**)&weights, mat.weights.size() * sizeof(double)))
    CHECK_CUDA(hipMalloc((void**)&cols, mat.cols.size() * sizeof(int)))
    CHECK_CUDA(hipMalloc((void**)&row_starts, mat.row_starts.size() * sizeof(int)))
    CHECK_CUDA(hipMemcpy(weights, mat.weights.data(), mat.weights.size() * sizeof(double), hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(cols, mat.cols.data(), mat.cols.size() * sizeof(int), hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(row_starts, mat.row_starts.data(), mat.row_starts.size() * sizeof(int), hipMemcpyHostToDevice))
    CHECK_CUSPARSE(hipsparseCreateCsr(&descr, mat.row_size(), mat.col_size(), mat.nnz(),
          row_starts, cols, weights,
          HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
          HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F))
  }
  ~CSRMatDev() {
    CHECK_CUSPARSE(hipsparseDestroySpMat(descr))
    CHECK_CUDA(hipFree(weights))
    CHECK_CUDA(hipFree(cols))
    CHECK_CUDA(hipFree(row_starts))
  }
  double *weights;
  int *cols;
  int *row_starts;
  hipsparseSpMatDescr_t descr;
};

struct DnVec {
  explicit DnVec(const std::vector<double>& v) : length(v.size()) {
    CHECK_CUDA(hipMalloc((void**)&data, length * sizeof(double)))
    CHECK_CUDA(hipMemcpy(data, v.data(), length * sizeof(double), hipMemcpyHostToDevice))
    CHECK_CUSPARSE(hipsparseCreateDnVec(&descr, length, data, HIP_R_64F))
  }
  explicit DnVec(size_t length) : length(length) {
    CHECK_CUDA(hipMalloc((void**)&data, length * sizeof(double)))
    CHECK_CUDA(hipMemset(data, 0, length * sizeof(double)))
    CHECK_CUSPARSE(hipsparseCreateDnVec(&descr, length, data, HIP_R_64F))
  }
  ~DnVec() {
    CHECK_CUSPARSE(hipsparseDestroyDnVec(descr))
    CHECK_CUDA(hipFree(data))
  }
  size_t size() const { return length; }
  size_t length;
  double *data;
  hipsparseDnVecDescr_t descr;
};

__global__ void sub(const double *src1, const double *src2, double *dst, size_t n) {
  const size_t stride = blockDim.x * gridDim.x;
  const size_t index = threadIdx.x + blockIdx.x * blockDim.x;
  for (size_t i = index; i < n; i += stride) {
    dst[i] = src1[i] - src2[i];
  }
}

__global__ void fma_sc(const double src1, const double *src2, double *dst, size_t n) {
  const size_t stride = blockDim.x * gridDim.x;
  const size_t index = threadIdx.x + blockIdx.x * blockDim.x;
  for (size_t i = index; i < n; i += stride) {
    dst[i] += src1 * src2[i];
  }
}

__global__ void fma4_sc(const double src1, const double *src2, const double *acc, double *dst, size_t n) {
  const size_t stride = blockDim.x * gridDim.x;
  const size_t index = threadIdx.x + blockIdx.x * blockDim.x;
  for (size_t i = index; i < n; i += stride) {
    dst[i] = src1 * src2[i] + acc[i];
  }
}

__global__ void accum_l2(double* buf_acc, const double* v, size_t n) {
  const size_t stride = blockDim.x * gridDim.x;
  const size_t index = threadIdx.x + blockIdx.x * blockDim.x;
  double result = 0.0;
  for (size_t i = index; i < n; i += stride) {
    result += v[i] * v[i];
  }
  buf_acc[index] = result;
}

__global__ void accum_l1(double* l1_acc, const double* y, size_t n) {
  const size_t stride = blockDim.x * gridDim.x;
  const size_t index = threadIdx.x + blockIdx.x * blockDim.x;
  double result = 0.0;
  for (size_t i = index; i < n; i += stride) {
    result += abs(y[i]);
  }
  l1_acc[index] = result;
}

void solve_impl(const Context& context, CSRMatDev& mat, const CSRMatDev& mat_tr, const DnVec& a, const DnVec& b) {
  double alpha = 1.0;
  double beta = 0.0;
  auto external_buffer = [&] {
    size_t bufsize_forward = 0, bufsize_backward = 0;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(context.handle.get(), HIPSPARSE_OPERATION_NON_TRANSPOSE,
          &alpha,
          mat.descr,
          a.descr,
          &beta,
          b.descr,
          HIP_R_64F,
          HIPSPARSE_SPMV_CSR_ALG1,
          &bufsize_forward))
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(context.handle.get(), HIPSPARSE_OPERATION_NON_TRANSPOSE,
          &alpha,
          mat_tr.descr,
          b.descr,
          &beta,
          a.descr,
          HIP_R_64F,
          HIPSPARSE_SPMV_CSR_ALG1,
          &bufsize_backward))
    const size_t bufsize = std::max(bufsize_forward, bufsize_backward);
    void *buf;
    CHECK_CUDA(hipMalloc(&buf, bufsize))
    return buf;
  }();
  auto spmv_non_trans = [&] (auto&& v_in, auto&& v_out) {
    CHECK_CUSPARSE(hipsparseSpMV(context.handle.get(), HIPSPARSE_OPERATION_NON_TRANSPOSE,
          &alpha,
          mat.descr,
          v_in.descr,
          &beta,
          v_out.descr,
          HIP_R_64F,
          HIPSPARSE_SPMV_CSR_ALG1,
          external_buffer))
  };
  auto spmv_trans = [&] (auto&& v_in, auto&& v_out) {
    CHECK_CUSPARSE(hipsparseSpMV(context.handle.get(), HIPSPARSE_OPERATION_NON_TRANSPOSE,
          &alpha,
          mat_tr.descr,
          v_in.descr,
          &beta,
          v_out.descr,
          HIP_R_64F,
          HIPSPARSE_SPMV_CSR_ALG1,
          external_buffer))
  };
  auto sub_vec = [&] (auto&& src1, auto&& src2, auto&& dst) {
    sub<<<1024, 256, 0, context.stream.get()>>>(src1.data, src2.data, dst.data, dst.size());
  };
  auto fma_sc_vec = [&] (double src1, auto&& src2, auto&& dst) {
    fma_sc<<<1024, 256, 0, context.stream.get()>>>(src1, src2.data, dst.data, dst.size());
  };
  auto fma4_sc_vec = [&] (double src1, auto&& src2, auto&& acc, auto&& dst) {
    fma4_sc<<<1024, 256, 0, context.stream.get()>>>(src1, src2.data, acc.data, dst.data, dst.size());
  };
  double* buf_acc;
  CHECK_CUDA(hipMallocManaged((void**)&buf_acc, 1024 * sizeof(double)))
  auto l1_norm = [&] (auto&& v) {
    accum_l1<<<8, 128, 0, context.stream.get()>>>(buf_acc, v.data, v.size()); 
    CHECK_CUDA(hipStreamSynchronize(context.stream.get()))
    double sum = 0.0;
    for (size_t i = 0; i < 1024; ++i) {
      sum += buf_acc[i];
    }
    return sum;
  };
  auto l2_norm = [&] (auto&& v) {
    accum_l2<<<8, 128, 0, context.stream.get()>>>(buf_acc, v.data, v.size()); 
    CHECK_CUDA(hipStreamSynchronize(context.stream.get()))
    double sum = 0.0;
    for (size_t i = 0; i < 1024; ++i) {
      sum += buf_acc[i];
    }
    return sum;
  };

  DnVec pa(b.size());
  spmv_non_trans(a, pa);
  DnVec r(b.size());
  sub_vec(b, pa, r);
  DnVec p(a.size());
  spmv_trans(r, p);
  DnVec s(a.size());
  CHECK_CUDA(hipMemcpyAsync(s.data, p.data, s.size() * sizeof(double), hipMemcpyDeviceToDevice, context.stream.get()))
  double old_s_norm = l2_norm(s);
  DnVec q(b.size());

  for (size_t i = 0; i < 300; ++i) {
    spmv_non_trans(p, q);
    const auto alpha = old_s_norm / l2_norm(q);
    fma_sc_vec(alpha, p, a);
    fma_sc_vec(-alpha, q, r);
    spmv_trans(r, s);
    const auto new_s_norm = l2_norm(s);
    if (i % 10 == 0) {
      double l1 = l1_norm(r) / r.size();
      std::cerr << i << " " << l1 << std::endl;
    }
    if (new_s_norm < 1.0) {
      break;
    }
    const auto beta = new_s_norm / old_s_norm;
    fma4_sc_vec(beta, p, s, p);
    old_s_norm = new_s_norm;
  }
  CHECK_CUDA(hipStreamSynchronize(context.stream.get()))
  hipFree(external_buffer);
  hipFree(buf_acc);
}

void solve(const Context& context, const CSRMat& mat, std::vector<double>& a, const std::vector<double>& b) {
  CSRMatDev dev_mat(mat);
  const auto dev_mat_tr = [&] {
    const auto mat_tr = transpose(mat);
    return CSRMatDev(mat_tr);
  }();
  DnVec dev_a(a);
  DnVec dev_b(b);
  solve_impl(context, dev_mat, dev_mat_tr, dev_a, dev_b);
  CHECK_CUDA(hipMemcpyAsync(a.data(), dev_a.data, a.size() * sizeof(double), hipMemcpyDeviceToHost, context.stream.get()))
}

int main(int argc, char** argv) {
  const auto param = parse_options(argc, argv);
  const auto config = load_config(param.config_path);
  const auto data_set = load_dataset(param.input_path);

  hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
  Context context;
  CHECK_CUSPARSE(hipsparseSetStream(context.handle.get(), context.stream.get()))

  PatternIndexer indexer(config.masks);
  size_t vec_len = indexer.pattern_size() + 3 + 1; // pattern, global(3), constant(1)
  std::vector<double> vec(vec_len);
  for (size_t mid = param.from; mid <= param.to; ++mid) {
    std::cerr << mid << std::endl;
    const size_t lower = mid - param.width + 1;
    const size_t upper = mid + param.width - 1;
    DataSet filtered;
    for (auto&& state : data_set) {
      size_t stone_count = state.stone_count();
      if (lower <= stone_count && stone_count <= upper) {
        filtered.push_back(state);
      }
    }
    const auto mat = generate_matrix(filtered, indexer);
    std::vector<double> scores;
    for (auto&& state : filtered) {
      scores.push_back(state.score);
    }
    scores.push_back(0.0); // L2 normalization
    solve(context, mat, vec, scores);
    std::ofstream ofs(param.output_path + "/weight_" + std::to_string(mid));
    for (auto&& w : vec) {
      ofs << w << "\n";
    }
  }
  return 0;
}
